#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cubicTex3D.cu>

#include "project_kernel_data.cu"

#define UPDATE(multiplier, vol_id, mat_id) do {\
    /* param. weight is set to 1.0f / (float)n_vols_at_curr_priority */\
    if (1.0f == tex3D(VOLUME(vol_id), px[vol_id], py[vol_id], pz[vol_id])) {\
        if (seg_at_alpha[vol_id][1] != 1.0f) {\
            printf("pos: {%f, %f, %f}. volume_%d == 1, seg_at_alpha[%d][1] != 1.0f\n", px[vol_id], py[vol_id], pz[vol_id], vol_id, vol_id);\
        }\
    }\
    area_density[(mat_id)] += (multiplier) * tex3D(VOLUME(vol_id), px[vol_id], py[vol_id], pz[vol_id]) * seg_at_alpha[vol_id][mat_id] * volume_normalization_factor[vol_id] * weight;\
} while (0)

#define GET_POSITION_FOR_VOL(vol_id) do {\
    /* Get the current sample point in the volume voxel-space. */\
    /* In CUDA, voxel centers are located at (xx.5, xx.5, xx.5), whereas SwVolume has voxel centers at integers. */\
    px[vol_id] = sx[vol_id] + alpha * rx[vol_id] - gVolumeEdgeMinPointX[vol_id];\
    py[vol_id] = sy[vol_id] + alpha * ry[vol_id] - gVolumeEdgeMinPointY[vol_id];\
    pz[vol_id] = sz[vol_id] + alpha * rz[vol_id] - gVolumeEdgeMinPointZ[vol_id];\
} while (0)

#define LOAG_SEGS_FOR_VOL_MAT(vol_id, mat_id) do {\
    seg_at_alpha[vol_id][mat_id] = round(cubicTex3D(SEG(vol_id, mat_id), px[vol_id], py[vol_id], pz[vol_id]));\
} while (0)

#if NUM_MATERIALS == 1
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
} while (0)
#elif NUM_MATERIALS == 2
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
} while (0)
#elif NUM_MATERIALS == 3
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
} while (0)
#elif NUM_MATERIALS == 4
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 3);\
} while (0)
#elif NUM_MATERIALS == 5
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 3);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 4);\
} while (0)
#elif NUM_MATERIALS == 6
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 3);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 4);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 5);\
} while (0)
#elif NUM_MATERIALS == 7
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 3);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 4);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 5);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 6);\
} while (0)
#elif NUM_MATERIALS == 8
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 3);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 4);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 5);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 6);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 7);\
} while (0)
#elif NUM_MATERIALS == 9
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 3);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 4);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 5);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 6);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 7);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 8);\
} while (0)
#elif NUM_MATERIALS == 10
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 3);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 4);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 5);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 6);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 7);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 8);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 9);\
} while (0)
#elif NUM_MATERIALS == 11
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 3);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 4);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 5);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 6);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 7);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 8);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 9);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 10);\
} while (0)
#elif NUM_MATERIALS == 12
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 3);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 4);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 5);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 6);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 7);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 8);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 9);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 10);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 11);\
} while (0)
#elif NUM_MATERIALS == 13
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 3);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 4);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 5);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 6);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 7);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 8);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 9);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 10);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 11);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 12);\
} while (0)
#elif NUM_MATERIALS == 14
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 0);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 1);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 2);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 3);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 4);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 5);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 6);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 7);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 8);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 9);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 10);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 11);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 12);\
    LOAG_SEGS_FOR_VOL_MAT(vol_id, 13);\
} while (0)
#else
#define LOAD_SEGS_FOR_VOL(vol_id) do {\
    fprintf(stderr, "NUM_MATERIALS not in [1, 14]");\
} while (0)
#endif

#if NUM_VOLUMES == 1
#define LOAD_SEGS_AT_ALPHA do {\
    if (do_trace[0]) { GET_POSITION_FOR_VOL(0); LOAD_SEGS_FOR_VOL(0); }\
} while (0)
#elif NUM_VOLUMES == 2
#define LOAD_SEGS_AT_ALPHA do {\
    if (do_trace[0]) { GET_POSITION_FOR_VOL(0); LOAD_SEGS_FOR_VOL(0); }\
    if (do_trace[1]) { GET_POSITION_FOR_VOL(1); LOAD_SEGS_FOR_VOL(1); }\
} while (0)
#elif NUM_VOLUMES == 3
#define LOAD_SEGS_AT_ALPHA do {\
    if (do_trace[0]) { GET_POSITION_FOR_VOL(0); LOAD_SEGS_FOR_VOL(0); }\
    if (do_trace[1]) { GET_POSITION_FOR_VOL(1); LOAD_SEGS_FOR_VOL(1); }\
    if (do_trace[2]) { GET_POSITION_FOR_VOL(2); LOAD_SEGS_FOR_VOL(2); }\
} while (0)
#else
#define LOAD_SEGS_AT_ALPHA do {\
    fprintf(stderr, "LOAD_SEGS_AT_ALPHA not supported for NUM_VOLUMES outside [1, 3]");\
} while (0)
#endif

#if NUM_MATERIALS == 1
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
} while (0)
#elif NUM_MATERIALS == 2
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
} while (0)
#elif NUM_MATERIALS == 3
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
} while (0)
#elif NUM_MATERIALS == 4
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
} while (0)
#elif NUM_MATERIALS == 5
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
} while (0)
#elif NUM_MATERIALS == 6
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
} while (0)
#elif NUM_MATERIALS == 7
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
} while (0)
#elif NUM_MATERIALS == 8
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
} while (0)
#elif NUM_MATERIALS == 9
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
} while (0)
#elif NUM_MATERIALS == 10
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
    UPDATE(multiplier, vol_id, 9);\
} while (0)
#elif NUM_MATERIALS == 11
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
    UPDATE(multiplier, vol_id, 9);\
    UPDATE(multiplier, vol_id, 10);\
} while (0)
#elif NUM_MATERIALS == 12
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
    UPDATE(multiplier, vol_id, 9);\
    UPDATE(multiplier, vol_id, 10);\
    UPDATE(multiplier, vol_id, 11);\
} while (0)
#elif NUM_MATERIALS == 13
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
    UPDATE(multiplier, vol_id, 9);\
    UPDATE(multiplier, vol_id, 10);\
    UPDATE(multiplier, vol_id, 11);\
    UPDATE(multiplier, vol_id, 12);\
} while (0)
#elif NUM_MATERIALS == 14
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
    UPDATE(multiplier, vol_id, 9);\
    UPDATE(multiplier, vol_id, 10);\
    UPDATE(multiplier, vol_id, 11);\
    UPDATE(multiplier, vol_id, 12);\
    UPDATE(multiplier, vol_id, 13);\
} while (0)
#else
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    fprintf(stderr, "NUM_MATERIALS not in [1, 14]");\
} while (0)
#endif

#if NUM_VOLUMES == 1
#define INTERPOLATE(multiplier) do {\
    if (do_trace[0] && (priority[0] == curr_priority)) { INTERPOLATE_FOR_VOL(multiplier, 0); }\
} while (0)
#elif NUM_VOLUMES == 2
#define INTERPOLATE(multiplier) do {\
    if (do_trace[0] && (priority[0] == curr_priority)) { INTERPOLATE_FOR_VOL(multiplier, 0); }\
    if (do_trace[1] && (priority[1] == curr_priority)) { INTERPOLATE_FOR_VOL(multiplier, 1); }\
} while (0)
#elif NUM_VOLUMES == 3
#define INTERPOLATE(multiplier) do {\
    if (do_trace[0] && (priority[0] == curr_priority)) { INTERPOLATE_FOR_VOL(multiplier, 0); }\
    if (do_trace[1] && (priority[1] == curr_priority)) { INTERPOLATE_FOR_VOL(multiplier, 1); }\
    if (do_trace[2] && (priority[2] == curr_priority)) { INTERPOLATE_FOR_VOL(multiplier, 2); }\
} while (0)
#else
#define INTERPOLATE(multiplier) do {\
    fprintf(stderr, "INTERPOLATE not supported for NUM_VOLUMES outside [1, 3]");\
} while (0)
#endif

#define CALCULATE_RAY_FOR_VOL(vol_id) do {\
    rx[vol_id] = u * rt_kinv[(9 * vol_id) + 0] + v * rt_kinv[(9 * vol_id) + 1] + rt_kinv[(9 * vol_id) + 2];\
    ry[vol_id] = u * rt_kinv[(9 * vol_id) + 3] + v * rt_kinv[(9 * vol_id) + 4] + rt_kinv[(9 * vol_id) + 5];\
    rz[vol_id] = u * rt_kinv[(9 * vol_id) + 6] + v * rt_kinv[(9 * vol_id) + 7] + rt_kinv[(9 * vol_id) + 8];\
    /* make the ray a unit vector */\
    float normFactor = 1.0f / sqrt((rx[vol_id] * rx[vol_id]) + (ry[vol_id] * ry[vol_id]) + (rz[vol_id] * rz[vol_id]));\
    rx[vol_id] *= normFactor;\
    ry[vol_id] *= normFactor;\
    rz[vol_id] *= normFactor;\
    \
    float tmp = 0.0f;\
    tmp += (rx[vol_id] * gVoxelElementSizeX[vol_id])*(rx[vol_id] * gVoxelElementSizeX[vol_id]);\
    tmp += (ry[vol_id] * gVoxelElementSizeY[vol_id])*(ry[vol_id] * gVoxelElementSizeY[vol_id]);\
    tmp += (rz[vol_id] * gVoxelElementSizeZ[vol_id])*(rz[vol_id] * gVoxelElementSizeZ[vol_id]);\
    volume_normalization_factor[vol_id] = sqrtf(tmp);\
} while (0)

#if NUM_VOLUMES == 1
#define CALCULATE_RAYS do {\
    CALCULATE_RAY_FOR_VOL(0);\
} while (0)
#elif NUM_VOLUMES == 2
#define CALCULATE_RAYS do {\
    CALCULATE_RAY_FOR_VOL(0);\
    CALCULATE_RAY_FOR_VOL(1);\
} while (0)
#elif NUM_VOLUMES == 3
#define CALCULATE_RAYS do {\
    CALCULATE_RAY_FOR_VOL(0);\
    CALCULATE_RAY_FOR_VOL(1);\
    CALCULATE_RAY_FOR_VOL(2);\
} while (0)
#else
#define CALCULATE_RAYS do {\
    fprintf(stderr, "CALCULATE_RAYS not supported for NUM_VOLUMES outside [1, 3]");\
} while (0)
#endif

#define CALCULATE_ALPHAS_FOR_VOL(i) do{\
    minAlpha[i] = 0;\
    maxAlpha[i] = INFINITY;\
    do_trace[i] = 1;\
\
    if (0.0f != rx[i]) {\
        float reci = 1.0f / rx[i];\
        float alpha0 = (gVolumeEdgeMinPointX[i] - sx[i]) * reci;\
        float alpha1 = (gVolumeEdgeMaxPointX[i] - sx[i]) * reci;\
        minAlpha[i] = fmin(alpha0, alpha1);\
        maxAlpha[i] = fmax(alpha0, alpha1);\
    } else if (gVolumeEdgeMinPointX[i] > sx[i] || sx[i] > gVolumeEdgeMaxPointX[i]) {\
        do_trace[i] = 0;\
    }\
\
    if (do_trace[i] && (0.0f != ry[i])) {\
        float reci = 1.0f / ry[i];\
        float alpha0 = (gVolumeEdgeMinPointY[i] - sy[i]) * reci;\
        float alpha1 = (gVolumeEdgeMaxPointY[i] - sy[i]) * reci;\
        minAlpha[i] = fmax(minAlpha[i], fmin(alpha0, alpha1));\
        maxAlpha[i] = fmin(maxAlpha[i], fmax(alpha0, alpha1));\
    } else if (gVolumeEdgeMinPointY[i] > sy[i] || sy[i] > gVolumeEdgeMaxPointY[i]) {\
        do_trace[i] = 0;\
    }\
\
    if (do_trace[i] && (0.0f != rz[i]))  {\
        float reci = 1.0f / rz[i];\
        float alpha0 = (gVolumeEdgeMinPointZ[i] - sz[i]) * reci;\
        float alpha1 = (gVolumeEdgeMaxPointZ[i] - sz[i]) * reci;\
        minAlpha[i] = fmax(minAlpha[i], fmin(alpha0, alpha1));\
        maxAlpha[i] = fmin(maxAlpha[i], fmax(alpha0, alpha1));\
    } else if (gVolumeEdgeMinPointZ > sz || sz > gVolumeEdgeMaxPointZ) {\
        do_trace[i] = 0;\
    }\
    globalMinAlpha = fmin(minAlpha[i], globalMinAlpha);\
    globalMaxAlpha = fmax(maxAlpha[i], globalMaxAlpha);\
} while (0)

#if NUM_VOLUMES == 1
#define CALCULATE_ALPHAS do {\
    CALCULATE_ALPHAS_FOR_VOL(0);\
} while (0)
#elif NUM_VOLUMES == 2
#define CALCULATE_ALPHAS do {\
    CALCULATE_ALPHAS_FOR_VOL(0);\
    CALCULATE_ALPHAS_FOR_VOL(1);\
} while (0)
#elif NUM_VOLUMES == 3
#define CALCULATE_ALPHAS do {\
    CALCULATE_ALPHAS_FOR_VOL(0);\
    CALCULATE_ALPHAS_FOR_VOL(1);\
    CALCULATE_ALPHAS_FOR_VOL(2);\
} while (0)
#else
#define CALCULATE_ALPHAS do {\
    fprintf(stderr, "CALCULATE_ALPHAS not supported for NUM_VOLUMES outside [1, 3]");\
} while (0)
#endif

#define GET_PRIORITY_AT_ALPHA do {\
    curr_priority = NUM_VOLUMES;\
    n_vols_at_curr_priority = 0;\
    for (int i = 0; i < NUM_VOLUMES; i++) {\
        if (0 == do_trace[i]) { continue; }\
        if ((alpha < minAlpha[i]) || (alpha > maxAlpha[i])) { continue; }\
        float any_seg = 0.0f;\
        for (int m = 0; m < NUM_MATERIALS; m++) {\
            any_seg += seg_at_alpha[i][m];\
            if (any_seg > 0.0f) { break; }\
        }\
        if (0.0f == any_seg) { continue; }\
\
        if (priority[i] < curr_priority) {\
            curr_priority = priority[i];\
            n_vols_at_curr_priority = 1;\
        } else if (priority[i] == curr_priority) {\
            n_vols_at_curr_priority ++;\
        }\
    }\
} while (0)

extern "C" {
    __global__  void projectKernel(
        int out_width, // width of the output image
        int out_height, // height of the output image
        float step,
        int *priority, // volumes with smaller priority-ID have higher priority when determining which volume we are in
        float *gVolumeEdgeMinPointX, // one value for each of the NUM_VOLUMES volumes
        float *gVolumeEdgeMinPointY,
        float *gVolumeEdgeMinPointZ,
        float *gVolumeEdgeMaxPointX,
        float *gVolumeEdgeMaxPointY,
        float *gVolumeEdgeMaxPointZ,
        float *gVoxelElementSizeX, // one value for each of the NUM_VOLUMES volumes
        float *gVoxelElementSizeY,
        float *gVoxelElementSizeZ,
        float *sx, // x-coordinate of source point for rays in world-space
        float *sy, // one value for each of the NUM_VOLUMES volumes
        float *sz,
        float *rt_kinv, // (NUM_VOLUMES, 3, 3) array giving the image-to-world-ray transform for each volume
        int n_bins, // the number of spectral bins
        float *energies, // 1-D array -- size is the n_bins. Units: [keV]
        float *pdf, // 1-D array -- probability density function over the energies
        float *absorb_coef_table, // flat [n_bins x NUM_MATERIALS] table that represents
                        // the precomputed get_absorption_coef values.
                        // index into the table as: table[bin * NUM_MATERIALS + mat]
        float *intensity, // flat array, with shape (out_height, out_width).
        float *photon_prob, // flat array, with shape (out_height, out_width).
        int offsetW,
        int offsetH)
    {
        // The output image has the following coordinate system, with cell-centered sampling.
        // y is along the fast axis (columns), x along the slow (rows).
        // Each point has NUM_MATERIALS elements at it.
        // 
        //      x -->
        //    y *---------------------------*
        //    | |                           |
        //    V |                           |
        //      |        output image       |
        //      |                           |
        //      |                           |
        //      *---------------------------*
        // 
        //
        int udx = threadIdx.x + (blockIdx.x + offsetW) * blockDim.x; // index into output image width
        int vdx = threadIdx.y + (blockIdx.y + offsetH) * blockDim.y; // index into output image height

        // if the current point is outside the output image, no computation needed
        if (udx >= out_width || vdx >= out_height)
            return;

        if ((0 == udx) && (0 == vdx)) {
            for (int i = 0; i < NUM_VOLUMES; i++) {
                printf(
                    "priority #%d: %d\n\tbounds #%d: [%f, %f], [%f, %f], [%f, %f]\n", 
                    i, priority[i], i, 
                    gVolumeEdgeMinPointX[i], gVolumeEdgeMaxPointX[i], 
                    gVolumeEdgeMinPointY[i], gVolumeEdgeMaxPointY[i], 
                    gVolumeEdgeMinPointZ[i], gVolumeEdgeMaxPointZ[i]
                );
            }
        }

        /*if ((0 == udx) && (0 == vdx)) {
            // test cubicTex3D around edges of volume1. volume1 is ones for slice [40:80, 40:60, 30:50]
            float x_min = 40.f;// + gVolumeEdgeMinPointX[1];
            float x_max = 80.f;// + gVolumeEdgeMinPointX[1];
            float y_min = 40.f;// + gVolumeEdgeMinPointY[1];
            float y_max = 60.f;// + gVolumeEdgeMinPointY[1];
            float z_min = 30.f;// + gVolumeEdgeMinPointZ[1];
            float z_max = 50.f;// + gVolumeEdgeMinPointZ[1];

            for (float x = x_min; x <= x_max; x += 0.5) {
                for (float y = y_min; y <= y_max; y += 0.5) {
                    for (float z = z_min; z <= z_max; z += 0.5) {
                        float seg_0_val = cubicTex3D(seg_1_0, x, y, z);
                        float seg_1_val = cubicTex3D(seg_1_1, x, y, z);

                        if (seg_0_val > 0.5f) {
                            printf("volume1, seg0 == %f > 0.5 in non-null section\n", seg_0_val);
                        }
                        if (seg_1_val < 0.5f) {
                            printf("volume1, seg1 == %f < 0.5 in non-null section: {%f, %f, %f}\n", seg_1_val, x, y, z);
                        }
                    }
                }
            }
        }*/

        // cell-centered sampling point corresponding to pixel index, in index-space.
        float u = (float) udx + 0.5;
        float v = (float) vdx + 0.5;

        // Vector in voxel-space along ray from source-point to pixel at [u,v] on the detector plane.
        float rx[NUM_VOLUMES];
        float ry[NUM_VOLUMES];
        float rz[NUM_VOLUMES];
        float volume_normalization_factor[NUM_VOLUMES];
        CALCULATE_RAYS;

        // calculate projections
        // Part 1: compute alpha value at entry and exit point of the volume on either side of the ray.
        // minAlpha: the distance from source point to volume entry point of the ray.
        // maxAlpha: the distance from source point to volume exit point of the ray.
        float minAlpha[NUM_VOLUMES];
        float maxAlpha[NUM_VOLUMES];
        int do_trace[NUM_VOLUMES]; // for each volume, whether or not to perform the ray-tracing
        float globalMinAlpha = INFINITY; // the smallest of all the minAlpha's
        float globalMaxAlpha = 0.0f; // the largest of all the maxAlpha's
        CALCULATE_ALPHAS;

        if ((600 == udx) && (400 == vdx)) {
            for (int i = 0; i < NUM_VOLUMES; i++) {
                printf("minAlpha[%d]=%f, maxAlpha[%d]=%f\n", i, minAlpha[i], i, maxAlpha[i]);
            }
            printf("globalMinAlpha=%f, globalMaxAlpha=%f\n", globalMinAlpha, globalMaxAlpha);
        }

        // we start not at the exact entry point 
        // => we can be sure to be inside the volume
        // (this is commented out intentionally, seemingly)
        //for (int i = 0; i < NUM_VOLUMES; i++) {
        //    minAlpha[i] += step * 0.5f;
        //}

        // Determine whether to do any ray-tracing at all.
        for (int i = 0; i < NUM_VOLUMES; i++) {
            if (do_trace[i]) { break; }
            else if ((NUM_VOLUMES - 1) == i) { return; }
        }
        
        // Part 2: Cast ray if it intersects the volume

        // material projection-output channels
        float area_density[NUM_MATERIALS]; 

        // initialize the projection-output to 0.
        for (int m = 0; m < NUM_MATERIALS; m++) {
            area_density[m] = 0.0f;
        }

        float px[NUM_VOLUMES]; // voxel-space point
        float py[NUM_VOLUMES];
        float pz[NUM_VOLUMES];
        float alpha; // distance along ray (alpha = globalMinAlpha + step * t)
        float boundary_factor; // factor to multiply at boundary
        int curr_priority; // the priority at the location
        int n_vols_at_curr_priority;//B[NUM_MATERIALS]; // how many volumes to consider at the location (for each material)
        float seg_at_alpha[NUM_VOLUMES][NUM_MATERIALS];

        for (alpha = globalMinAlpha; alpha < globalMaxAlpha; alpha += step) {
            LOAD_SEGS_AT_ALPHA; // initializes p{x,y,z}[...] and seg_at_alpha[...][...]
            GET_PRIORITY_AT_ALPHA;
            if (0 == n_vols_at_curr_priority) {
                // Outside the bounds of all volumes to trace. Assume nominal density of air is 0.0f.
                // Thus, we don't need to add to area_density
                ;
            } else {
                float weight = 1.0f / ((float)n_vols_at_curr_priority);

                // For the entry boundary, multiply by 0.5. That is, for the initial interpolated value,
                // only a half step-size is considered in the computation. For the second-to-last interpolation
                // point, also multiply by 0.5, since there will be a final step at the globalMaxAlpha boundary.
                boundary_factor = ((alpha <= globalMinAlpha) || (alpha + step >= globalMaxAlpha)) ? 0.5f : 1.0f;

                INTERPOLATE(boundary_factor);
            }
        }

        // Scaling by step
        for (int m = 0; m < NUM_MATERIALS; m++) {
            area_density[m] *= step;
        }

        // Last segment of the line
        if (area_density[0] > 0.0f) {
            alpha -= step;
            float lastStepsize = globalMaxAlpha - alpha;

            LOAD_SEGS_AT_ALPHA; // TODO: are this line and the next (both macros) necessary?
            GET_PRIORITY_AT_ALPHA;
            if (0 == n_vols_at_curr_priority) {
                // Outside the bounds of all volumes to trace. Assume nominal density of air is 0.0f.
                // Thus, we don't need to add to area_density
                ;
            } else {
                float weight = 1.0f / ((float)n_vols_at_curr_priority);

                // Scaled last step interpolation (something weird?)
                INTERPOLATE(lastStepsize);
            }
        }

        // Convert to centimeters
        for (int m = 0; m < NUM_MATERIALS; m++) {
            area_density[m] /= 10.0f;
        }

        /* Up to this point, we have accomplished the original projectKernel functionality.
         * The next steps to do are combining the forward_projections dictionary-ization and 
         * the mass_attenuation computation
         */

        // forward_projections dictionary-ization is implicit.

        // flat index to pixel in *intensity and *photon_prob
        int img_dx = (udx * out_height) + vdx;

        // zero-out intensity and photon_prob
        intensity[img_dx] = 0;
        photon_prob[img_dx] = 0;

        // MASS ATTENUATION COMPUTATION

        /**
         * EXPLANATION OF THE PHYSICS/MATHEMATICS
         * 
         *      The mass attenuation coefficient (found in absorb_coef_table) is: \mu / \rho, where
         * \mu is the linear attenuation coefficient, and \rho is the mass density.  \mu has units of
         * inverse length, and \rho has units of mass/volume, so the mass attenuation coefficient has
         * units of [cm^2 / g]
         *      area_density[m] is the product of [linear distance of the ray through material 'm'] and 
         * [density of the material].  Accordingly, area_density[m] has units of [g / cm^2].
         *
         * The mass attenuation code uses the Beer-Lambert law:
         *
         *      I = I_{0} exp[-(\mu / \rho) * \rho * d]
         *
         * where I_{0} is the initial intensity, (\mu / \rho) is the mass attenuation coefficient, 
         * \rho is the density, and d is the length of the ray passing through the material.  Note 
         * that the product (\rho * d), also known as the 'area density' is the quantity area_density[m].
         *      Because we are attenuating multiple materials, the exponent that we use for the 
         * Beer-Lambert law is the sum of the (\mu_{mat} / \rho_{mat}) * (\rho_{mat} * d_{mat}) for
         * each material 'mat'.
         *
         *      The above explains the calculation up to and including 
         *              '____ = expf(-1 * beer_lambert_exp)',
         * but does not yet explain the remaining calculation.  The remaining calculation serves to 
         * approximate the workings of a pixel in the dectector:
         *      
         *      pixelReading = \sum_{E} attenuatedBeamStrength[E] * E * p(E)
         *
         * where attenuatedBeamStrength follows the Beer-Lambert law as above, E is the energies of
         * the spectrum, and p(E) is the PDF of the spectrum.
         *      Note also that the Beer-Lambert law deals with the quantity 'intensity', which is 
         * related to the power transmitted through [unit area perpendicular to the direction of travel].
         * Since the intensities mentioned in the Beer-Lambert law are proportional to 1/[unit area], we
         * can replace the "intensity" calcuation with simply the energies involved.  Later conversion to 
         * other physical quanities can be done outside of the kernel.
         */
        for (int bin = 0; bin < n_bins; bin++) {
            float beer_lambert_exp = 0.0f;
            for (int m = 0; m < NUM_MATERIALS; m++) {
                beer_lambert_exp += area_density[m] * absorb_coef_table[bin * NUM_MATERIALS + m];
            }
            float photon_prob_tmp = expf(-1.f * beer_lambert_exp) * pdf[bin]; // dimensionless value

            photon_prob[img_dx] += photon_prob_tmp;
            intensity[img_dx] += energies[bin] * photon_prob_tmp; // units: [keV] per unit photon to hit the pixel
        }

        return;
    }
}
    
