#include "hip/hip_runtime.h"

#include <stdio.h>
#include <cubicTex3D.cu>

#ifndef NUM_MATERIALS
#define NUM_MATERIALS 14
#endif

#define _seg(n) seg_##n
#define seg(n) _seg(n)

//  channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> seg(0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> seg(1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> seg(2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> seg(3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> seg(4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> seg(5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> seg(6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> seg(7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> seg(8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> seg(9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> seg(10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> seg(11)
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> seg(12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> seg(13);
#endif

#define UPDATE(multiplier, n) ({\
    output[idx + n] += (multiplier) * tex3D(volume, px, py, pz) * round(cubicTex3D(seg(n), px, py, pz));\
})

#if NUM_MATERIALS == 1
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
})
#elif NUM_MATERIALS == 2
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
})
#elif NUM_MATERIALS == 3
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
})
#elif NUM_MATERIALS == 4
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 3);\
})
#elif NUM_MATERIALS == 5
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 3);\
    UPDATE(multiplier, 4);\
})
#elif NUM_MATERIALS == 6
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
})  
#elif NUM_MATERIALS == 7
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
})
#elif NUM_MATERIALS == 8
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
})
#elif NUM_MATERIALS == 9
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
})
#elif NUM_MATERIALS == 10
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
    UPDATE(multiplier, 9);\
})
#elif NUM_MATERIALS == 11
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
    UPDATE(multiplier, 9);\
    UPDATE(multiplierl, 10);\
})
#elif NUM_MATERIALS == 12
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
    UPDATE(multiplier, 9);\
    UPDATE(multiplier, 10);\
    UPDATE(multiplier, 11);\
})
#elif NUM_MATERIALS == 13
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
    UPDATE(multiplier, 9);\
    UPDATE(multiplier, 10);\
    UPDATE(multiplier, 11);\
    UPDATE(multiplier, 12);\
})
#elif NUM_MATERIALS == 14
#define INTERPOLATE(multiplier) ({\
    UPDATE(multiplier, 0);\
    UPDATE(multiplier, 1);\
    UPDATE(multiplier, 2);\
    UPDATE(multiplier, 4);\
    UPDATE(multiplier, 5);\
    UPDATE(multiplier, 6);\
    UPDATE(multiplier, 7);\
    UPDATE(multiplier, 8);\
    UPDATE(multiplier, 9);\
    UPDATE(multiplier, 10);\
    UPDATE(multiplier, 11);\
    UPDATE(multiplier, 12);\
    UPDATE(multiplier, 13);\
})
#else
#define INTERPOLATE(multiplier) (fprintf("NUM_MATERIALS not in [1, 14]"))
#endif


// the CT volume (used to be tex_density)
texture<float, 3, hipReadModeElementType> volume;

extern "C" {
    __global__  void projectKernel(
        int out_width, // width of the output image
        int out_height, // height of the output image
        float step,
        float gVolumeEdgeMinPointX,
        float gVolumeEdgeMinPointY,
        float gVolumeEdgeMinPointZ,
        float gVolumeEdgeMaxPointX,
        float gVolumeEdgeMaxPointY,
        float gVolumeEdgeMaxPointZ,
        float gVoxelElementSizeX,
        float gVoxelElementSizeY,
        float gVoxelElementSizeZ,
        float sx, // x-coordinate of source point for rays in world-space
        float sy,
        float sz,
        float* gInvARmatrix, // (3, 3) array giving the image-to-world-ray transform.
        float* output, // flat array, with shape (out_height, out_width, NUM_MATERIALS).
        int offsetW,
        int offsetH)
    {
        int widx = threadIdx.x + (blockIdx.x + offsetW) * blockDim.x; // index into output width
        int hidx = threadIdx.y + (blockIdx.y + offsetH) * blockDim.y; // index into output height

        // if the current point is outside the output image, no computation needed
        if (widx >= out_width || hidx >= out_height)
            return;

        // flat index to first material in output "channel". 
        // So (idx + m) gets you the pixel for material index m in [0, NUM_MATERIALS)
        int idx = widx * (out_height * NUM_MATERIALS) + hidx * NUM_MATERIALS; 

        // image-space point corresponding to pixel
        float u = (float) widx + 0.5;
        float v = (float) hidx + 0.5;

        // vector along world-space ray from source-point to pixel on the image plane
        float rx = u * gInvARmatrix[0] + v * gInvARmatrix[1] + gInvARmatrix[2];
        float ry = u * gInvARmatrix[3] + v * gInvARmatrix[4] + gInvARmatrix[5];
        float rz = u * gInvARmatrix[6] + v * gInvARmatrix[7] + gInvARmatrix[8];

        // make the ray a unit-vector
        float normFactor = 1.0f / (sqrt((rx * rx) + (ry * ry) + (rz * rz)));
        rx *= normFactor;
        ry *= normFactor;
        rz *= normFactor;

        // calculate projections
        // Part 1: compute alpha value at entry and exit point of the volume on either side of the ray.
        // minAlpha: the distance from source point to volume entry point of the ray.
        // maxAlpha: the distance from source point to volume exit point of the ray.
        float minAlpha, maxAlpha;
        minAlpha = 0;
        maxAlpha = INFINITY;

        if (0.0f != rx)
        {
            float reci = 1.0f / rx;
            float alpha0 = (gVolumeEdgeMinPointX - sx) * reci;
            float alpha1 = (gVolumeEdgeMaxPointX - sx) * reci;
            minAlpha = fmin(alpha0, alpha1);
            maxAlpha = fmax(alpha0, alpha1);
        }
        else if (gVolumeEdgeMinPointX > sx || sx > gVolumeEdgeMaxPointX)
        {
            return;
        }

        if (0.0f != ry)
        {
            float reci = 1.0f / ry;
            float alpha0 = (gVolumeEdgeMinPointY - sy) * reci;
            float alpha1 = (gVolumeEdgeMaxPointY - sy) * reci;
            minAlpha = fmax(minAlpha, fmin(alpha0, alpha1));
            maxAlpha = fmin(maxAlpha, fmax(alpha0, alpha1));
        }
        else if (gVolumeEdgeMinPointY > sy || sy > gVolumeEdgeMaxPointY)
        {
            return;
        }

        if (0.0f != rz)
        {
            float reci = 1.0f / rz;
            float alpha0 = (gVolumeEdgeMinPointZ - sz) * reci;
            float alpha1 = (gVolumeEdgeMaxPointZ - sz) * reci;
            minAlpha = fmax(minAlpha, fmin(alpha0, alpha1));
            maxAlpha = fmin(maxAlpha, fmax(alpha0, alpha1));
        }
        else if (gVolumeEdgeMinPointZ > sz || sz > gVolumeEdgeMaxPointZ)
        {
            return;
        }

        // we start not at the exact entry point 
        // => we can be sure to be inside the volume
        // (this is commented out intentionally, seemingly)
        //minAlpha += step * 0.5f;
        
        // Part 2: Cast ray if it intersects the volume

        // Trapezoidal rule (interpolating function = piecewise linear func)
        float px, py, pz; // world-space point
        int t; // number of steps along ray
        float alpha; // distance along ray (alpha = minAlpha + step * t)
        float boundary_factor; // factor to multiply at the boundary.

        // initialize the output to 0.
        for (int m = 0; m < NUM_MATERIALS; m++) {
            output[idx + m] = 0;
        }

        // Sample the points along the ray at the entrance boundary of the volume and the mid segments.
        for (t = 0, alpha = minAlpha; alpha < maxAlpha; t++, alpha += step)
        {
            // Get the current sample point in the volume world-space.
            // In CUDA, voxel centeras are located at (xx.5, xx.5, xx.5), whereas SwVolume has voxel centers at integers.
            px = sx + alpha * rx + 0.5;
            py = sy + alpha * ry + 0.5;
            pz = sz + alpha * rz - gVolumeEdgeMinPointZ;

            /* For the entry boundary, multiply by 0.5 (this is the i == 0 check). That is, for the initial interpolated value, 
             * only a half step-size is considered in the computation.
             * For the second-to-last interpolation point, also multiply by 0.5, since there will be a final step at the maxAlpha boundary.
             */ 
            boundary_factor = (t == 0 || alpha + step >= maxAlpha) ? 0.5 : 1.0;

            // Perform the interpolation. This involves the variables: output, idx, px, py, pz, and volume. 
            // It is done for each segmentation.
            INTERPOLATE(boundary_factor);
        }

        // Scaling by step;
        output[idx] *= step;

        // Last segment of the line
        if (output[idx] > 0.0f ) {
            alpha -= step;
            float lastStepsize = maxAlpha - alpha;

            // scaled last step interpolation (something weird?)
            INTERPOLATE(0.5 * lastStepsize);

            // The last segment of the line integral takes care of the varying length.
            px = sx + alpha * rx + 0.5;
            py = sy + alpha * ry + 0.5;
            pz = sz + alpha * rz - gVolumeEdgeMinPointZ;

            // interpolation
            INTERPOLATE(0.5 * lastStepsize);
        }

        // normalize output value to world coordinate system units
        for (int m = 0; m < NUM_MATERIALS; m++) {
            output[idx + m] *= sqrt((rx * gVoxelElementSizeX)*(rx * gVoxelElementSizeX) + (ry * gVoxelElementSizeY)*(ry * gVoxelElementSizeY) + (rz * gVoxelElementSizeZ)*(rz * gVoxelElementSizeZ));
        }
    
        return;
    }
}
    
