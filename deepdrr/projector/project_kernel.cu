#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cubicTex3D.cu>

#include "project_kernel_data.cu"

#define UPDATE(multiplier, vol_id, mat_id) do {\
    area_density[(mat_id)] += (multiplier) * tex3D(VOLUME(vol_id), px, py, pz) * round(cubicTex3D(SEG(vol_id, mat_id), px, py, pz));\
    /*area_density[(mat_id)] += (multiplier) * tex3D(VOLUME(0), px, py, pz) * round(cubicTex3D(SEG(vol_id, mat_id), px, py, pz));*/\
} while (0)

#if NUM_MATERIALS == 1
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
} while (0)
#elif NUM_MATERIALS == 2
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
} while (0)
#elif NUM_MATERIALS == 3
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
} while (0)
#elif NUM_MATERIALS == 4
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
} while (0)
#elif NUM_MATERIALS == 5
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
} while (0)
#elif NUM_MATERIALS == 6
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
} while (0)
#elif NUM_MATERIALS == 7
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
} while (0)
#elif NUM_MATERIALS == 8
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
} while (0)
#elif NUM_MATERIALS == 9
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
} while (0)
#elif NUM_MATERIALS == 10
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
    UPDATE(multiplier, vol_id, 9);\
} while (0)
#elif NUM_MATERIALS == 11
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
    UPDATE(multiplier, vol_id, 9);\
    UPDATE(multiplier, vol_id, 10);\
} while (0)
#elif NUM_MATERIALS == 12
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
    UPDATE(multiplier, vol_id, 9);\
    UPDATE(multiplier, vol_id, 10);\
    UPDATE(multiplier, vol_id, 11);\
} while (0)
#elif NUM_MATERIALS == 13
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
    UPDATE(multiplier, vol_id, 9);\
    UPDATE(multiplier, vol_id, 10);\
    UPDATE(multiplier, vol_id, 11);\
    UPDATE(multiplier, vol_id, 12);\
} while (0)
#elif NUM_MATERIALS == 14
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    UPDATE(multiplier, vol_id, 0);\
    UPDATE(multiplier, vol_id, 1);\
    UPDATE(multiplier, vol_id, 2);\
    UPDATE(multiplier, vol_id, 3);\
    UPDATE(multiplier, vol_id, 4);\
    UPDATE(multiplier, vol_id, 5);\
    UPDATE(multiplier, vol_id, 6);\
    UPDATE(multiplier, vol_id, 7);\
    UPDATE(multiplier, vol_id, 8);\
    UPDATE(multiplier, vol_id, 9);\
    UPDATE(multiplier, vol_id, 10);\
    UPDATE(multiplier, vol_id, 11);\
    UPDATE(multiplier, vol_id, 12);\
    UPDATE(multiplier, vol_id, 13);\
} while (0)
#else
#define INTERPOLATE_FOR_VOL(multiplier, vol_id) do {\
    fprintf(stderr, "NUM_MATERIALS not in [1, 14]");\
} while (0)
#endif

#define CALCULATE_RAY_FOR_VOL(vol_id) do {\
    rx[vol_id] = u * rt_kinv[(9 * vol_id) + 0] + v * rt_kinv[(9 * vol_id) + 1] + rt_kinv[(9 * vol_id) + 2];\
    ry[vol_id] = u * rt_kinv[(9 * vol_id) + 3] + v * rt_kinv[(9 * vol_id) + 4] + rt_kinv[(9 * vol_id) + 5];\
    rz[vol_id] = u * rt_kinv[(9 * vol_id) + 6] + v * rt_kinv[(9 * vol_id) + 7] + rt_kinv[(9 * vol_id) + 8];\
    /* make the ray a unit vector */\
    float normFactor = 1.0f / sqrt((rx[vol_id] * rx[vol_id]) + (ry[vol_id] * ry[vol_id]) + (rz[vol_id] * rz[vol_id]));\
    rx[vol_id] *= normFactor;\
    ry[vol_id] *= normFactor;\
    rz[vol_id] *= normFactor;\
} while (0)

#if NUM_VOLUMES == 1
#define CALCULATE_RAYS do {\
    CALCULATE_RAY_FOR_VOL(0);\
} while (0)
#elif NUM_VOLUMES == 2
#define CALCULATE_RAYS do {\
    CALCULATE_RAY_FOR_VOL(0);\
    CALCULATE_RAY_FOR_VOL(1);\
} while (0)
#elif NUM_VOLUMES == 3
#define CALCULATE_RAYS do {\
    CALCULATE_RAY_FOR_VOL(0);\
    CALCULATE_RAY_FOR_VOL(1);\
    CALCULATE_RAY_FOR_VOL(2);\
} while (0)
#else
#define CALCULATE_RAYS do {\
    fprintf(stderr, "CALCULATE_RAYS not supported for NUM_VOLUMES outside [1, 3]");\
} while (0)
#endif

#define CALCULATE_ALPHAS_FOR_VOL(i) do{\
    minAlpha[i] = 0;\
    maxAlpha[i] = INFINITY;\
    do_trace[i] = 1;\
\
    if (0.0f != rx[i]) {\
        float reci = 1.0f / rx[i];\
        float alpha0 = (gVolumeEdgeMinPointX[i] - sx[i]) * reci;\
        float alpha1 = (gVolumeEdgeMaxPointX[i] - sx[i]) * reci;\
        minAlpha[i] = fmin(alpha0, alpha1);\
        maxAlpha[i] = fmax(alpha0, alpha1);\
    } else if (gVolumeEdgeMinPointX[i] > sx[i] || sx[i] > gVolumeEdgeMaxPointX[i]) {\
        do_trace[i] = 0;\
    }\
\
    if (do_trace[i] && (0.0f != ry[i])) {\
        float reci = 1.0f / ry[i];\
        float alpha0 = (gVolumeEdgeMinPointY[i] - sy[i]) * reci;\
        float alpha1 = (gVolumeEdgeMaxPointY[i] - sy[i]) * reci;\
        minAlpha[i] = fmax(minAlpha[i], fmin(alpha0, alpha1));\
        maxAlpha[i] = fmin(maxAlpha[i], fmax(alpha0, alpha1));\
    } else if (gVolumeEdgeMinPointY[i] > sy[i] || sy[i] > gVolumeEdgeMaxPointY[i]) {\
        do_trace[i] = 0;\
    }\
\
    if (do_trace[i] && (0.0f != rz[i]))  {\
        float reci = 1.0f / rz[i];\
        float alpha0 = (gVolumeEdgeMinPointZ[i] - sz[i]) * reci;\
        float alpha1 = (gVolumeEdgeMaxPointZ[i] - sz[i]) * reci;\
        minAlpha[i] = fmax(minAlpha[i], fmin(alpha0, alpha1));\
        maxAlpha[i] = fmin(maxAlpha[i], fmax(alpha0, alpha1));\
    } else if (gVolumeEdgeMinPointZ > sz || sz > gVolumeEdgeMaxPointZ) {\
        do_trace[i] = 0;\
    }\
} while (0)

#if NUM_VOLUMES == 1
#define CALCULATE_ALPHAS do {\
    CALCULATE_ALPHAS_FOR_VOL(0);\
} while (0)
#elif NUM_VOLUMES == 2
#define CALCULATE_ALPHAS do {\
    CALCULATE_ALPHAS_FOR_VOL(0);\
    CALCULATE_ALPHAS_FOR_VOL(1);\
} while (0)
#elif NUM_VOLUMES == 3
#define CALCULATE_ALPHAS do {\
    CALCULATE_ALPHAS_FOR_VOL(0);\
    CALCULATE_ALPHAS_FOR_VOL(1);\
    CALCULATE_ALPHAS_FOR_VOL(2);\
} while (0)
#else
#define CALCULATE_ALPHAS do {\
    fprintf(stderr, "CALCULATE_ALPHAS not supported for NUM_VOLUMES outside [1, 3]");\
} while (0)
#endif

#define RAY_TRACE_FOR_VOL(i) do {\
    /* Trapezoidal rule (interpolating function = piecewise linear func) */\
    float px, py, pz; /* voxel-space point */\
    int t; /* number of steps along ray */\
    float alpha; /* distance along ray (alpha = minAlpha + step * t) */\
    float boundary_factor; /* factor to multiply at the boundary. */\
\
    /* Sample the points along the ray at the entrance boundary of the volume and the mid segments. */\
    for (t = 0, alpha = minAlpha[i]; alpha < maxAlpha[i]; t++, alpha += step)\
    {\
        /* Get the current sample point in the volume voxel-space. */\
        /* In CUDA, voxel centeras are located at (xx.5, xx.5, xx.5), whereas SwVolume has voxel centers at integers. */\
        px = sx[i] + alpha * rx[i] - gVolumeEdgeMinPointX[i];\
        py = sy[i] + alpha * ry[i] - gVolumeEdgeMinPointY[i];\
        pz = sz[i] + alpha * rz[i] - gVolumeEdgeMinPointZ[i];\
\
        /* For the entry boundary, multiply by 0.5 (this is the t == 0 check). That is, for the initial interpolated value, */\
        /* only a half step-size is considered in the computation. */\
        /* For the second-to-last interpolation point, also multiply by 0.5, since there will be a final step at the maxAlpha boundary.*/\
        boundary_factor = (t == 0 || alpha + step >= maxAlpha[i]) ? 0.5 : 1.0;\
\
        /* Perform the interpolation. This involves the variables: area_density, idx, px, py, pz, and volume. */\
        /* It is done for each segmentation. */\
        INTERPOLATE_FOR_VOL(boundary_factor, i);\
    }\
\
    /* Scaling by step; */\
    for (int m = 0; m < NUM_MATERIALS; m++) {\
        area_density[m] *= step;\
    }\
\
    /* Last segment of the line */\
    if (area_density[0] > 0.0f) {\
        alpha -= step;\
        float lastStepsize = maxAlpha[i] - alpha;\
        /* scaled last step interpolation (something weird?) */\
        INTERPOLATE_FOR_VOL(0.5 * lastStepsize, i);\
        /* The last segment of the line integral takes care of the varying length. */\
        px = sx[i] + alpha * rx[i] - gVolumeEdgeMinPointX[i];\
        py = sy[i] + alpha * ry[i] - gVolumeEdgeMinPointY[i];\
        pz = sz[i] + alpha * rz[i] - gVolumeEdgeMinPointZ[i];\
        /* interpolation */\
        INTERPOLATE_FOR_VOL(0.5 * lastStepsize, i);\
    }\
\
    /* normalize output value to world coordinate system units */\
    for (int m = 0; m < NUM_MATERIALS; m++) {\
        area_density[m] *= sqrt((rx[i] * gVoxelElementSizeX[i])*(rx[i] * gVoxelElementSizeX[i]) + (ry[i] * gVoxelElementSizeY[i])*(ry[i] * gVoxelElementSizeY[i]) + (rz[i] * gVoxelElementSizeZ[i])*(rz[i] * gVoxelElementSizeZ[i]));\
        \
        total_area_density[m] += area_density[m];\
    }\
} while (0)

#if NUM_VOLUMES == 1
#define RAY_TRACE do {\
    RAY_TRACE_FOR_VOL(0);\
} while (0)
#elif NUM_VOLUMES == 2
#define RAY_TRACE do {\
    RAY_TRACE_FOR_VOL(0);\
    RAY_TRACE_FOR_VOL(1);\
} while (0)
#elif NUM_VOLUMES == 3
#define RAY_TRACE do {\
    RAY_TRACE_FOR_VOL(0);\
    RAY_TRACE_FOR_VOL(1);\
    RAY_TRACE_FOR_VOL(2);\
} while (0)
#else
#define RAY_TRACE do {\
    fprintf(stderr, "RAY_TRACE not supported for NUM_VOLUMES outside [1, 3]");\
} while (0)
#endif

extern "C" {
    __global__  void projectKernel(
        int out_width, // width of the output image
        int out_height, // height of the output image
        float step,
        float *gVolumeEdgeMinPointX, // one value for each of the NUM_VOLUMES volumes
        float *gVolumeEdgeMinPointY,
        float *gVolumeEdgeMinPointZ,
        float *gVolumeEdgeMaxPointX,
        float *gVolumeEdgeMaxPointY,
        float *gVolumeEdgeMaxPointZ,
        float *gVoxelElementSizeX, // one value for each of the NUM_VOLUMES volumes
        float *gVoxelElementSizeY,
        float *gVoxelElementSizeZ,
        float *sx, // x-coordinate of source point for rays in world-space
        float *sy, // one value for each of the NUM_VOLUMES volumes
        float *sz,
        float *rt_kinv, // (NUM_VOLUMES, 3, 3) array giving the image-to-world-ray transform for each volume
        int n_bins, // the number of spectral bins
        float *energies, // 1-D array -- size is the n_bins. Units: [keV]
        float *pdf, // 1-D array -- probability density function over the energies
        float *absorb_coef_table, // flat [n_bins x NUM_MATERIALS] table that represents
                        // the precomputed get_absorption_coef values.
                        // index into the table as: table[bin * NUM_MATERIALS + mat]
        float *intensity, // flat array, with shape (out_height, out_width).
        float *photon_prob, // flat array, with shape (out_height, out_width).
        int offsetW,
        int offsetH)
    {
        // The output image has the following coordinate system, with cell-centered sampling.
        // y is along the fast axis (columns), x along the slow (rows).
        // Each point has NUM_MATERIALS elements at it.
        // 
        //      x -->
        //    y *---------------------------*
        //    | |                           |
        //    V |                           |
        //      |        output image       |
        //      |                           |
        //      |                           |
        //      *---------------------------*
        // 
        //
        int udx = threadIdx.x + (blockIdx.x + offsetW) * blockDim.x; // index into output image width
        int vdx = threadIdx.y + (blockIdx.y + offsetH) * blockDim.y; // index into output image height

        // if the current point is outside the output image, no computation needed
        if (udx >= out_width || vdx >= out_height)
            return;

        // cell-centered sampling point corresponding to pixel index, in index-space.
        float u = (float) udx + 0.5;
        float v = (float) vdx + 0.5;

        // Vector in voxel-space along ray from source-point to pixel at [u,v] on the detector plane.
        float rx[NUM_VOLUMES];
        float ry[NUM_VOLUMES];
        float rz[NUM_VOLUMES];
        CALCULATE_RAYS;

        // calculate projections
        // Part 1: compute alpha value at entry and exit point of the volume on either side of the ray.
        // minAlpha: the distance from source point to volume entry point of the ray.
        // maxAlpha: the distance from source point to volume exit point of the ray.
        float minAlpha[NUM_VOLUMES];
        float maxAlpha[NUM_VOLUMES];
        int do_trace[NUM_VOLUMES]; // for each volume, whether or not to perform the ray-tracing
        CALCULATE_ALPHAS;

        // we start not at the exact entry point 
        // => we can be sure to be inside the volume
        // (this is commented out intentionally, seemingly)
        //for (int i = 0; i < NUM_VOLUMES; i++) {
        //    minAlpha[i] += step * 0.5f;
        //}

        // Determine whether to do any ray-tracing at all. 
        // Use [out_width] as variable because it is no longer needed
        for (out_width = 0; out_width < NUM_VOLUMES; out_width++) {
            if (do_trace[out_width]) { break; }
        }
        if (NUM_VOLUMES == out_width) {
            return;
        }
        
        // Part 2: Cast ray if it intersects the volume

        // material projection-output channels
        float total_area_density[NUM_MATERIALS]; // total over all volumes
        float area_density[NUM_MATERIALS]; // temp storage for each volume 

        // initialize the projection-output to 0.
        for (int m = 0; m < NUM_MATERIALS; m++) {
            total_area_density[m] = 0;
            area_density[m] = 0;
        }

        RAY_TRACE;

        // Convert to centimeters
        for (int m = 0; m < NUM_MATERIALS; m++) {
            total_area_density[m] /= 10.0f;
        }

        /* Up to this point, we have accomplished the original projectKernel functionality.
         * The next steps to do are combining the forward_projections dictionary-ization and 
         * the mass_attenuation computation
         */

        // forward_projections dictionary-ization is implicit.

        // flat index to pixel in *intensity and *photon_prob
        int img_dx = (udx * out_height) + vdx;

        // zero-out intensity and photon_prob
        intensity[img_dx] = 0;
        photon_prob[img_dx] = 0;

        // MASS ATTENUATION COMPUTATION

        /**
         * EXPLANATION OF THE PHYSICS/MATHEMATICS
         * 
         *      The mass attenuation coefficient (found in absorb_coef_table) is: \mu / \rho, where
         * \mu is the linear attenuation coefficient, and \rho is the mass density.  \mu has units of
         * inverse length, and \rho has units of mass/volume, so the mass attenuation coefficient has
         * units of [cm^2 / g]
         *      area_density[m] is the product of [linear distance of the ray through material 'm'] and 
         * [density of the material].  Accordingly, area_density[m] has units of [g / cm^2].
         *
         * The mass attenuation code uses the Beer-Lambert law:
         *
         *      I = I_{0} exp[-(\mu / \rho) * \rho * d]
         *
         * where I_{0} is the initial intensity, (\mu / \rho) is the mass attenuation coefficient, 
         * \rho is the density, and d is the length of the ray passing through the material.  Note 
         * that the product (\rho * d), also known as the 'area density' is the quantity area_density[m].
         *      Because we are attenuating multiple materials, the exponent that we use for the 
         * Beer-Lambert law is the sum of the (\mu_{mat} / \rho_{mat}) * (\rho_{mat} * d_{mat}) for
         * each material 'mat'.
         *
         *      The above explains the calculation up to and including 
         *              '____ = expf(-1 * beer_lambert_exp)',
         * but does not yet explain the remaining calculation.  The remaining calculation serves to 
         * approximate the workings of a pixel in the dectector:
         *      
         *      pixelReading = \sum_{E} attenuatedBeamStrength[E] * E * p(E)
         *
         * where attenuatedBeamStrength follows the Beer-Lambert law as above, E is the energies of
         * the spectrum, and p(E) is the PDF of the spectrum.
         *      Note also that the Beer-Lambert law deals with the quantity 'intensity', which is 
         * related to the power transmitted through [unit area perpendicular to the direction of travel].
         * Since the intensities mentioned in the Beer-Lambert law are proportional to 1/[unit area], we
         * can replace the "intensity" calcuation with simply the energies involved.  Later conversion to 
         * other physical quanities can be done outside of the kernel.
         */
        for (int bin = 0; bin < n_bins; bin++) {
            float beer_lambert_exp = 0.0f;
            for (int m = 0; m < NUM_MATERIALS; m++) {
                beer_lambert_exp += total_area_density[m] * absorb_coef_table[bin * NUM_MATERIALS + m];
            }
            float photon_prob_tmp = expf(-1.f * beer_lambert_exp) * pdf[bin]; // dimensionless value

            photon_prob[img_dx] += photon_prob_tmp;
            intensity[img_dx] += energies[bin] * photon_prob_tmp; // units: [keV] per unit photon to hit the pixel
        }

        return;
    }
}
    
